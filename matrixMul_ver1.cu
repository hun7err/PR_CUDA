#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include "genmatrix.h"

#include <hip/hip_runtime.h>

__global__ void matrixMulSingleBlock(float *C, float *A, float *B, int width)
{
	int blocksPerDim = (int)ceil((float)width / (float)blockDim.x);
	int row;
	int col;
	float C_local = 0;

	for(int i = 0; i < blocksPerDim; i++)
	{
		for(int j = 0; j < blocksPerDim; j++)
		{
			row = threadIdx.y + i * blockDim.y;
			col = threadIdx.x + j * blockDim.x;

			if(row < width && col < width)
			{
				for(int k = 0; k < width; k++)
				{

					C_local += A[row * width + k] * B[k * width + col];
				}
			
				C[row * width + col] = C_local;
			}
		}
	}
}

static float totalTime = 0.0f;

int performSingleBlockTest(dim3 block_size, int width)
{
	hipError_t error;

	float *A = (float*)malloc(width*width*sizeof(float));
	float *B = (float*)malloc(width*width*sizeof(float));

	generateTestMatrix(A, width);
	generateTestMatrix(B, width);
	
	float *C = (float*)malloc(width*width*sizeof(float));
	memset(C, 0, width*width*sizeof(float));

	float *A_d, *B_d, *C_d;

	error = hipMalloc((void**)&A_d, width*width*sizeof(float));

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not allocate memory on the device for matrix A: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMalloc((void**)&B_d, width*width*sizeof(float));

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not allocate memory on the device for matrix B: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMalloc((void**)&C_d, width*width*sizeof(float));

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not allocate memory on the device for matrix C: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMemcpy(A_d, A, width*width*sizeof(float), hipMemcpyHostToDevice);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not copy data from host to device: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMemcpy(B_d, B, width*width*sizeof(float), hipMemcpyHostToDevice);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not copy data from host to device: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
    }

	error = hipEventRecord(start, NULL);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not record start event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	for(int current_test = 0; current_test < TEST_COUNT; current_test++)
	{
		matrixMulSingleBlock<<<dim3((int)ceil((float)width/(float)block_size.x), (int)ceil((float)width/(float)block_size.y)),block_size>>>(C_d, A_d, B_d, width);

		hipDeviceSynchronize();
	}

	error = hipEventRecord(stop, NULL);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not record stop event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipEventSynchronize(stop);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not synchronize with stop event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
	}

	totalTime = 0.0f;
	error = hipEventElapsedTime(&totalTime, start, stop);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not calculate elapsed time: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
	}

	float msecPerMatrixMul = totalTime / (float)TEST_COUNT;
    double flopsPerMatrixMul = 2.0 * (double)width * (double)width * (double)width;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
	
	printf("%dx%d\t%dx%d\t%.3f\t%.2f\n", width, width, block_size.x, block_size.y, msecPerMatrixMul, gigaFlops);

	error = hipMemcpy(C, C_d, width*width*sizeof(float), hipMemcpyDeviceToHost);
	
	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not copy data from device to host: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(C_d);
	hipFree(B_d);
	hipFree(A_d);

	free(C);
	free(B);
	free(A);

	return 0;
}

void performSingleBlockTests()
{
	int matrixSizes[] = { 32, 64, 128 };
	dim3 blockSizes[] = {dim3(8,8), dim3(16,16), dim3(22,22), dim3(32,32)};
	
	for(int i = 1; i < sizeof(blockSizes)/sizeof(dim3); i += 2)
	{
		for(int j = 0; j < sizeof(matrixSizes)/sizeof(int); j++)
		{
			performSingleBlockTest(blockSizes[i], matrixSizes[j]);
		}
	}

	for(int i = 0; i < sizeof(blockSizes)/sizeof(dim3); i++)
	{
		for(int j = 0; j < sizeof(matrixSizes)/sizeof(int); j += 2)
		{
			performSingleBlockTest(blockSizes[i], matrixSizes[j]);
		}
	}

	hipDeviceReset();
	//
}