#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

template <int BLOCK_SIZE> __global__ void matrixMulSharedMemBasic(float *C, float *A, float *B, int width)
{
	int a_start = width * BLOCK_SIZE * blockIdx.y;

	__shared__ float A_shared[BLOCK_SIZE*BLOCK_SIZE];
	__shared__ float B_shared[BLOCK_SIZE*BLOCK_SIZE];

	float C_local = 0.0f;

	int b_index = BLOCK_SIZE * blockIdx.x;

	for(int a_index = a_start; a_index < a_start + width; a_index += BLOCK_SIZE)
	{
		A_shared[threadIdx.y * blockDim.x + threadIdx.x] = A[a_index + width * threadIdx.y + threadIdx.x];
		B_shared[threadIdx.y * blockDim.x + threadIdx.x] = B[b_index + width * threadIdx.y + threadIdx.x];

		__syncthreads();

		for(int k = 0; k < BLOCK_SIZE; k++)
		{
			C_local += A_shared[threadIdx.y * BLOCK_SIZE + k] * B_shared[k * BLOCK_SIZE + threadIdx.x];
		}

		__syncthreads();

		b_index += BLOCK_SIZE * width;
	}

	int c_start = width * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x;
	C[c_start + width * threadIdx.y + threadIdx.x] = C_local;
}

void performSharedMemTest(void)
{
	float A[] = {1.0f,2.0f,3.0f,4.0f,5.0f,6.0f,7.0f,8.0f,9.0f,10.0f,11.0f,12.0f,13.0f,14.0f,15.0f,16.0f};
	float B[] = {17.0f,18.0f,19.0f,20.0f,21.0f,22.0f,23.0f,24.0f,25.0f,26.0f,27.0f,28.0f,29.0f,30.0f,31.0f,32.0f};

	int width = 4;

	float *C = (float*)malloc(width*width*sizeof(float));
	memset(C, 0.0f, width*width*sizeof(float));

	float *A_d, *B_d, *C_d;
	hipMalloc((void**)&A_d, width*width*sizeof(float));
	hipMalloc((void**)&B_d, width*width*sizeof(float));
	hipMalloc((void**)&C_d, width*width*sizeof(float));

	hipMemcpy(A_d, A, width*width*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, width*width*sizeof(float), hipMemcpyHostToDevice);
	hipMemset(C_d, 0, width*width*sizeof(float));

	matrixMulSharedMemBasic<2><<<dim3(2,2), dim3(2,2)>>>(C_d, A_d, B_d, width);
	hipMemcpy(C, C_d, width*width*sizeof(float), hipMemcpyDeviceToHost);

	printf("[\n");
	for(int i = 0; i < 4; i++)
	{
		printf("\t[");
		for(int j = 0; j < 4; j++)
		{
			printf("%f,", C[i*4+j]);
		}
		printf("],\n");
	}
	printf("]\n\n");

	hipFree(C_d);
	hipFree(B_d);
	hipFree(A_d);

	free(C);
}
