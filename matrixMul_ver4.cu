#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "genmatrix.h"

#include <hip/hip_runtime.h>

template <int BLOCK_SIZE> __global__ void matrixMulSharedMemPrefetch(float *C, float *A, float *B, int width) // doda� sprawdzanie < width
{
	int a_start = width * BLOCK_SIZE * blockIdx.y, a_offset,
		b_start = BLOCK_SIZE * blockIdx.x, b_offset;

	__shared__ float A_shared[BLOCK_SIZE*BLOCK_SIZE];
	__shared__ float B_shared[BLOCK_SIZE*BLOCK_SIZE];

	float C_local = 0.0f;

	float a_prefetched = A[a_start + threadIdx.y * width + threadIdx.x],
			b_prefetched = B[b_start + threadIdx.y * width + threadIdx.x];

	for(int index = 0; index < gridDim.x;) // r�wnie dobrze mog�oby by� gridDim.y bo s� r�wne
	{
		++index;

		a_offset = index * BLOCK_SIZE;
		b_offset = index * BLOCK_SIZE * width;

		A_shared[threadIdx.y * blockDim.x + threadIdx.x] = a_prefetched;
		B_shared[threadIdx.y * blockDim.x + threadIdx.x] = b_prefetched;
		
		__syncthreads();

		if(index < gridDim.x)
		{
			a_prefetched = A[a_start + a_offset + threadIdx.y * width + threadIdx.x];
			b_prefetched = B[b_start + b_offset + threadIdx.y * width + threadIdx.x];
		}

		for(int k = 0; k < BLOCK_SIZE; k++)
		{
			C_local += A_shared[threadIdx.y * BLOCK_SIZE + k] * B_shared[k * BLOCK_SIZE + threadIdx.x];
		}

		__syncthreads(); // bariera synchronizacyjna, czekamy a� wszystkie w�tki w bloku oblicz� wynik cz�stkowy

		if(index * BLOCK_SIZE >= width)
			break;
	}
	
	int c_start = blockIdx.y * width * BLOCK_SIZE,
		c_offset = blockIdx.x * BLOCK_SIZE;
	C[c_start + c_offset + width * threadIdx.y + threadIdx.x] = C_local;
}

static float totalTime = 0.0f;
#define TEST_COUNT 42 // inside joke

int performImprovedSharedMemTest(dim3 block_size, int width)
{
	hipError_t error;

	float *A = (float*)malloc(width*width*sizeof(float));
	float *B = (float*)malloc(width*width*sizeof(float));

	generateTestMatrix(A, width);
	generateTestMatrix(B, width);
	
	float *C = (float*)malloc(width*width*sizeof(float));
	memset(C, 0, width*width*sizeof(float));

	float *A_d, *B_d, *C_d;

	error = hipMalloc((void**)&A_d, width*width*sizeof(float));

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not allocate memory on the device for matrix A: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMalloc((void**)&B_d, width*width*sizeof(float));

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not allocate memory on the device for matrix B: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMalloc((void**)&C_d, width*width*sizeof(float));

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not allocate memory on the device for matrix C: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMemcpy(A_d, A, width*width*sizeof(float), hipMemcpyHostToDevice);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not copy data from host to device: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMemcpy(B_d, B, width*width*sizeof(float), hipMemcpyHostToDevice);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not copy data from host to device: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
    }

	error = hipEventRecord(start, NULL);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not record start event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	int grid_side = (int)ceil((float)width/(float)block_size.x);

	for(int current_test = 0; current_test < TEST_COUNT; current_test++)
	{
		switch(block_size.x)
		{
			case 8:
				matrixMulSharedMemPrefetch<8><<<dim3(grid_side, grid_side), block_size>>>(C_d, A_d, B_d, width);
			break;
			case 16:
				matrixMulSharedMemPrefetch<16><<<dim3(grid_side, grid_side), block_size>>>(C_d, A_d, B_d, width);
			break;
			case 22:
				matrixMulSharedMemPrefetch<22><<<dim3(grid_side, grid_side), block_size>>>(C_d, A_d, B_d, width);
			break;
			case 32:
				matrixMulSharedMemPrefetch<32><<<dim3(grid_side, grid_side), block_size>>>(C_d, A_d, B_d, width);
			break;
		}
		
		hipDeviceSynchronize();
	}

	error = hipEventRecord(stop, NULL);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not record stop event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipEventSynchronize(stop);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not synchronize with stop event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
	}
	
	totalTime = 0.0f;
	error = hipEventElapsedTime(&totalTime, start, stop);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not calculate elapsed time: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
	}

	float msecPerMatrixMul = totalTime / (float)TEST_COUNT;
    double flopsPerMatrixMul = 2.0 * (double)width * (double)width * (double)width;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);

	printf("%dx%d\t%dx%d\t%dx%d\t%.3f\t%.2f\n", width, width, block_size.x, block_size.y, grid_side, grid_side, msecPerMatrixMul, gigaFlops);

	error = hipMemcpy(C, C_d, width*width*sizeof(float), hipMemcpyDeviceToHost);
	
	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not copy data from device to host: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(C_d);
	hipFree(B_d);
	hipFree(A_d);

	free(C);
	free(B);
	free(A);

	return 0;
}

void performImprovedSharedMemTests(void)
{
	srand((unsigned int)time(NULL));

	dim3 blockSizes[] = { dim3(8,8), dim3(16,16), dim3(22,22), dim3(32,32)};
	int matrixSizes[] = { 32, 64, 128 };

	for(int i = 0; i < sizeof(matrixSizes)/sizeof(int); i++)
	{
		for(int j = 0; j < sizeof(blockSizes)/sizeof(dim3); j++)
		{
			performImprovedSharedMemTest(blockSizes[j], matrixSizes[i]);
		}
	}
}
