#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

template <int BLOCK_SIZE> __global__ void matrixMulSharedMemPrefetch(float *C, float *A, float *B, int width) // doda� sprawdzanie < width
{
	int a_start = width * BLOCK_SIZE * blockIdx.y, a_offset,
		b_start = BLOCK_SIZE * blockIdx.x, b_offset;

	__shared__ float A_shared[BLOCK_SIZE*BLOCK_SIZE];
	__shared__ float B_shared[BLOCK_SIZE*BLOCK_SIZE];

	float C_local = 0.0f;

	float a_prefetched = A[a_start + threadIdx.y * width + threadIdx.x],
			b_prefetched = B[b_start + threadIdx.y * width + threadIdx.x];

	for(int index = 0; index < gridDim.x;) // r�wnie dobrze mog�oby by� gridDim.y bo s� r�wne
	{
		++index;

		a_offset = index * BLOCK_SIZE;
		b_offset = index * BLOCK_SIZE * width;

		A_shared[threadIdx.y * blockDim.x + threadIdx.x] = a_prefetched;
		B_shared[threadIdx.y * blockDim.x + threadIdx.x] = b_prefetched;
		
		__syncthreads();

		if(index < gridDim.x)
		{
			a_prefetched = A[a_start + a_offset + threadIdx.y * width + threadIdx.x];
			b_prefetched = B[b_start + b_offset + threadIdx.y * width + threadIdx.x];
		}

		for(int k = 0; k < BLOCK_SIZE; k++)
		{
			C_local += A_shared[threadIdx.y * BLOCK_SIZE + k] * B_shared[k * BLOCK_SIZE + threadIdx.x];
		}

		__syncthreads(); // bariera synchronizacyjna, czekamy a� wszystkie w�tki w bloku oblicz� wynik cz�stkowy

		if(index * BLOCK_SIZE >= width)
			break;
	}
	
	int c_start = blockIdx.y * width * BLOCK_SIZE,
		c_offset = blockIdx.x * BLOCK_SIZE;
	C[c_start + c_offset + width * threadIdx.y + threadIdx.x] = C_local;
}

void performImprovedSharedMemTest(void)
{
	float A[] = {1.0f,2.0f,3.0f,4.0f,5.0f,6.0f,7.0f,8.0f,9.0f,10.0f,11.0f,12.0f,13.0f,14.0f,15.0f,16.0f};
	float B[] = {17.0f,18.0f,19.0f,20.0f,21.0f,22.0f,23.0f,24.0f,25.0f,26.0f,27.0f,28.0f,29.0f,30.0f,31.0f,32.0f};

	int width = 4;

	float *C = (float*)malloc(width*width*sizeof(float));
	memset(C, 0.0f, width*width*sizeof(float));

	float *A_d, *B_d, *C_d;
	hipMalloc((void**)&A_d, width*width*sizeof(float));
	hipMalloc((void**)&B_d, width*width*sizeof(float));
	hipMalloc((void**)&C_d, width*width*sizeof(float));

	hipMemcpy(A_d, A, width*width*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, width*width*sizeof(float), hipMemcpyHostToDevice);
	hipMemset(C_d, 0, width*width*sizeof(float));

	matrixMulSharedMemPrefetch<2><<<dim3(2,2), dim3(2,2)>>>(C_d, A_d, B_d, width);
	hipMemcpy(C, C_d, width*width*sizeof(float), hipMemcpyDeviceToHost);

	printf("[\n");
	for(int i = 0; i < 4; i++)
	{
		printf("\t[");
		for(int j = 0; j < 4; j++)
		{
			printf("%f,", C[i*4+j]);
		}
		printf("],\n");
	}
	printf("]\n\n");

	hipFree(C_d);
	hipFree(B_d);
	hipFree(A_d);

	free(C);
}
