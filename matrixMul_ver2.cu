#include "hip/hip_runtime.h"
#define WIN32
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include "genmatrix.h"

#include <hip/hip_runtime.h>

 __global__ void matrixMulMultiBlock(float *C, float *A, float *B, int width)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	float C_local = 0;

	if(row < width && col < width)
	{
		for(int k = 0; k < width; k++)
			C_local += A[row*width+k] * B[k*width+col];
	
		C[row*width+col] = C_local;
	}
}

static float totalTime = 0.0f;
#define TEST_COUNT 300

int performMultiBlockTest(dim3 block_size, int width)
{
	printf("Block size (%d,%d) matrix width %d\n", block_size.x, block_size.y, width);

	hipError_t error;

	float *A = (float*)malloc(width*width*sizeof(float));
	float *B = (float*)malloc(width*width*sizeof(float));

	generateTestMatrix(A, width);
	generateTestMatrix(B, width);
	
	float *C = (float*)malloc(width*width*sizeof(float));
	memset(C, 0, width*width*sizeof(float));

	float *A_d, *B_d, *C_d;

	error = hipMalloc((void**)&A_d, width*width*sizeof(float));

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not allocate memory on the device for matrix A: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMalloc((void**)&B_d, width*width*sizeof(float));

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not allocate memory on the device for matrix B: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMalloc((void**)&C_d, width*width*sizeof(float));

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not allocate memory on the device for matrix C: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMemcpy(A_d, A, width*width*sizeof(float), hipMemcpyHostToDevice);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not copy data from host to device: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMemcpy(B_d, B, width*width*sizeof(float), hipMemcpyHostToDevice);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not copy data from host to device: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
    }

	error = hipEventRecord(start, NULL);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not record start event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	for(int current_test = 0; current_test < TEST_COUNT; current_test++)
	{
		matrixMulMultiBlock<<<dim3((int)ceil((float)width/(float)block_size.x), (int)ceil((float)width/(float)block_size.y)),block_size>>>(C_d, A_d, B_d, width);
	}

	error = hipEventRecord(stop, NULL);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not record stop event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipEventSynchronize(stop);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not synchronize with stop event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
	}
	
	totalTime = 0.0f;
	error = hipEventElapsedTime(&totalTime, start, stop);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not calculate elapsed time: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
	}

	float msecPerMatrixMul = totalTime / (float)TEST_COUNT;
    double flopsPerMatrixMul = 2.0 * (double)width * (double)width * (double)width;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);

	printf("Performance: %.2f GFlop/s, time: %.3f ms\n", gigaFlops, msecPerMatrixMul);

	error = hipMemcpy(C, C_d, width*width*sizeof(float), hipMemcpyDeviceToHost);
	
	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not copy data from device to host: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(C_d);
	hipFree(B_d);
	hipFree(A_d);

	free(C);
	free(B);
	free(A);

	return 0;
}

void performMultiBlockTests(void)
{
	srand((unsigned int)time(NULL));

	dim3 blockSizes[] = { dim3(8,8), dim3(16,16), dim3(22,22), dim3(32,32)};
	int matrixSizes[] = { 32, 64, 128 };

	for(int i = 0; i < sizeof(matrixSizes)/sizeof(int); i++)
	{
		printf("+++ %dx%d matrix +++\n", matrixSizes[i], matrixSizes[i]);

		for(int j = 0; j < sizeof(blockSizes)/sizeof(dim3); j++)
		{
			printf("%dx%d block\n", blockSizes[i].x, blockSizes[i].y);

			performMultiBlockTest(blockSizes[i], matrixSizes[i]);
		}
	}
}