#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "genmatrix.h"

#include <hip/hip_runtime.h>

template <int BLOCK_SIZE, int threadElemsPerDim> __global__ void matrixMulSharedMemPrefetchMultipleElements(float *C, float *A, float *B, int width) // sprawdzi� czemu to nie dzia�a
{
	int a_start = width * BLOCK_SIZE * threadElemsPerDim * blockIdx.y, a_offset, // pocz�tek wiersza z A przez kt�ry b�dziemy si� przeiterowywa�
		b_start = BLOCK_SIZE * threadElemsPerDim * blockIdx.x, b_offset;		 // pocz�tek kolumny z B przez kt�r� b�dziemy si� przeiterowywa�

	__shared__ float A_shared[BLOCK_SIZE*threadElemsPerDim*BLOCK_SIZE*threadElemsPerDim];
	__shared__ float B_shared[BLOCK_SIZE*threadElemsPerDim*BLOCK_SIZE*threadElemsPerDim];

	//float C_local = 0.0f;
	float C_local[threadElemsPerDim*threadElemsPerDim];

	float a_prefetched[threadElemsPerDim*threadElemsPerDim],
			b_prefetched[threadElemsPerDim*threadElemsPerDim];

	int row, col;

	for(row = 0; row < threadElemsPerDim; row++)
	{
		for(col = 0; col < threadElemsPerDim; col++)
		{
			a_prefetched[row*threadElemsPerDim+col] = A[a_start + (threadIdx.y + row) * width + threadIdx.x + col];
			b_prefetched[row*threadElemsPerDim+col] = B[b_start + (threadIdx.y + row) * width + threadIdx.x + col];
			C_local[row*threadElemsPerDim+col] = 0.0f;
		}
	}
	// up: domniemanie poprawno�ci

	for(int index = 0; index < gridDim.x;) // r�wnie dobrze mog�oby by� gridDim.y bo s� r�wne
	{
		++index;

		a_offset = index * BLOCK_SIZE * threadElemsPerDim;
		b_offset = index * BLOCK_SIZE * threadElemsPerDim * width;

		// <ok>
		for(row = 0; row < threadElemsPerDim; row++)
		{
			for(col = 0; col < threadElemsPerDim; col++)
			{
				A_shared[(threadIdx.y + row) * blockDim.x * threadElemsPerDim + threadIdx.x + col] = a_prefetched[row*threadElemsPerDim+col];
				B_shared[(threadIdx.y + row) * blockDim.x * threadElemsPerDim + threadIdx.x + col] = b_prefetched[row*threadElemsPerDim+col];
			}
		}
		// </ok>

		__syncthreads(); // bariera synchronizacyjna, czekamy a� wszystkie w�tki w bloku wype�ni� pami�� wsp�dzielon�

		for(row = 0; row < threadElemsPerDim; row++)
		{
			for(col = 0; col < threadElemsPerDim; col++)
			{
				if(index < gridDim.x)
				{
					a_prefetched[row*threadElemsPerDim+col] = A[a_start + a_offset + (threadIdx.y + row) * width + threadIdx.x + col];
					b_prefetched[row*threadElemsPerDim+col] = B[b_start + b_offset + (threadIdx.y + row) * width + threadIdx.x + col];
				}
			
				for(int k = 0; k < BLOCK_SIZE*threadElemsPerDim; k++)
				{
					C_local[row*threadElemsPerDim+col] += A_shared[(threadIdx.y + row) * BLOCK_SIZE * threadElemsPerDim + k] * B_shared[k * BLOCK_SIZE * threadElemsPerDim + threadIdx.x + col];
				}
			}
		}

		__syncthreads(); // bariera synchronizacyjna, czekamy a� wszystkie w�tki w bloku oblicz� wynik cz�stkowy

		if(index * BLOCK_SIZE * threadElemsPerDim >= width)
			break;
	}

	int c_start = blockIdx.y * width * BLOCK_SIZE * threadElemsPerDim,
		c_offset = blockIdx.x * BLOCK_SIZE * threadElemsPerDim;
	for(row = 0; row < threadElemsPerDim; row++)
	{
		for(col = 0; col < threadElemsPerDim; col++)
		{
			C[c_start + c_offset + width * (threadIdx.y + row) + threadIdx.x + col] = C_local[row*threadElemsPerDim+col];
		}
	}
}

static float totalTime = 0.0f;
#define TEST_COUNT 42 // inside joke
#define THREAD_ELEMENTS_PER_DIM 2

int performImprovedSharedMemMultipleElemsTest(dim3 block_size, int width)
{
	hipError_t error;

	float *A = (float*)malloc(width*width*sizeof(float));
	float *B = (float*)malloc(width*width*sizeof(float));

	generateTestMatrix(A, width);
	generateTestMatrix(B, width);
	
	float *C = (float*)malloc(width*width*sizeof(float));
	memset(C, 0, width*width*sizeof(float));

	float *A_d, *B_d, *C_d;

	error = hipMalloc((void**)&A_d, width*width*sizeof(float));

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not allocate memory on the device for matrix A: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMalloc((void**)&B_d, width*width*sizeof(float));

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not allocate memory on the device for matrix B: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMalloc((void**)&C_d, width*width*sizeof(float));

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not allocate memory on the device for matrix C: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMemcpy(A_d, A, width*width*sizeof(float), hipMemcpyHostToDevice);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not copy data from host to device: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMemcpy(B_d, B, width*width*sizeof(float), hipMemcpyHostToDevice);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not copy data from host to device: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
    }

	error = hipEventRecord(start, NULL);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not record start event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	int grid_side = (int)ceil((float)width/(float)block_size.x/(float)THREAD_ELEMENTS_PER_DIM);

	for(int current_test = 0; current_test < TEST_COUNT; current_test++)
	{
		switch(block_size.x)
		{
			case 8:
				matrixMulSharedMemPrefetchMultipleElements<8, THREAD_ELEMENTS_PER_DIM><<<dim3(grid_side, grid_side), block_size>>>(C_d, A_d, B_d, width);
			break;
			case 16:
				matrixMulSharedMemPrefetchMultipleElements<16, THREAD_ELEMENTS_PER_DIM><<<dim3(grid_side, grid_side), block_size>>>(C_d, A_d, B_d, width);
			break;
			case 22:
				matrixMulSharedMemPrefetchMultipleElements<22, THREAD_ELEMENTS_PER_DIM><<<dim3(grid_side, grid_side), block_size>>>(C_d, A_d, B_d, width);
			break;
			case 32:
				matrixMulSharedMemPrefetchMultipleElements<32, THREAD_ELEMENTS_PER_DIM><<<dim3(grid_side, grid_side), block_size>>>(C_d, A_d, B_d, width);
			break;
		}

		hipDeviceSynchronize();
	}

	error = hipEventRecord(stop, NULL);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not record stop event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipEventSynchronize(stop);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not synchronize with stop event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
	}
	
	totalTime = 0.0f;
	error = hipEventElapsedTime(&totalTime, start, stop);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not calculate elapsed time: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
	}

	float msecPerMatrixMul = totalTime / (float)TEST_COUNT;
    double flopsPerMatrixMul = 2.0 * (double)width * (double)width * (double)width;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);

	printf("%dx%d\t%dx%d\t%dx%d\t%dx%d\t%.3f\t%.2f\n", width, width, block_size.x * THREAD_ELEMENTS_PER_DIM, block_size.y * THREAD_ELEMENTS_PER_DIM, block_size.x, block_size.y, grid_side, grid_side, msecPerMatrixMul, gigaFlops);

	error = hipMemcpy(C, C_d, width*width*sizeof(float), hipMemcpyDeviceToHost);
	
	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not copy data from device to host: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(C_d);
	hipFree(B_d);
	hipFree(A_d);

	free(C);
	free(B);
	free(A);

	return 0;
}

void performImprovedSharedMemMultipleElemsTests(void)
{
	srand((unsigned int)time(NULL));

	dim3 blockSizes[] = { dim3(8/THREAD_ELEMENTS_PER_DIM,8/THREAD_ELEMENTS_PER_DIM), dim3(16/THREAD_ELEMENTS_PER_DIM,16/THREAD_ELEMENTS_PER_DIM), dim3(22/THREAD_ELEMENTS_PER_DIM,22/THREAD_ELEMENTS_PER_DIM), dim3(32/THREAD_ELEMENTS_PER_DIM,32/THREAD_ELEMENTS_PER_DIM) };
	int matrixSizes[] = { 32, 64, 128 };

	for(int i = 0; i < sizeof(matrixSizes)/sizeof(int); i++)
	{
		for(int j = 0; j < sizeof(blockSizes)/sizeof(dim3); j++)
		{
			performImprovedSharedMemMultipleElemsTest(blockSizes[j], matrixSizes[i]);
		}
	}
}

/*void performImprovedSharedMemMultipleElemsTest(void)
{
	float A[] = {1.0f,2.0f,3.0f,4.0f,5.0f,6.0f,7.0f,8.0f,9.0f,10.0f,11.0f,12.0f,13.0f,14.0f,15.0f,16.0f};
	float B[] = {17.0f,18.0f,19.0f,20.0f,21.0f,22.0f,23.0f,24.0f,25.0f,26.0f,27.0f,28.0f,29.0f,30.0f,31.0f,32.0f};

	int width = 4;

	float *C = (float*)malloc(width*width*sizeof(float));
	memset(C, 0.0f, width*width*sizeof(float));

	float *A_d, *B_d, *C_d;
	hipMalloc((void**)&A_d, width*width*sizeof(float));
	hipMalloc((void**)&B_d, width*width*sizeof(float));
	hipMalloc((void**)&C_d, width*width*sizeof(float));

	hipMemcpy(A_d, A, width*width*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, width*width*sizeof(float), hipMemcpyHostToDevice);
	hipMemset(C_d, 0, width*width*sizeof(float));

	const unsigned int threadElemsPerDim = 2;
	const unsigned int blockSideOrig = 2;
	const unsigned int blockSide = (int)ceil((float)blockSideOrig/(float)threadElemsPerDim);

	//matrixMulSharedMemPrefetchMultipleElements<blockSide,threadElemsPerDim> <<< dim3(2,2), dim3(blockSide, blockSide) >>>(C_d, A_d, B_d, width); // niby na tej linii jest jaki� b��d ._.
	printf("grid size: %dx%d\n", (int)ceil((float)width/(float)blockSideOrig), (int)ceil((float)width/(float)blockSideOrig));
	printf("elements per thread: a %dx%d submatrix\n", threadElemsPerDim, threadElemsPerDim);
	printf("new block size: %dx%d (original %dx%d)\n", blockSide, blockSide, blockSideOrig, blockSideOrig);
	matrixMulSharedMemPrefetchMultipleElements<1,2><<<dim3((int)ceil((float)width/(float)blockSideOrig),(int)ceil((float)width/(float)blockSideOrig)),dim3(blockSide,blockSide)>>>(C_d, A_d, B_d, width); // niby na tej linii jest jaki� b��d ._.
	
	hipMemcpy(C, C_d, width*width*sizeof(float), hipMemcpyDeviceToHost);

	printf("[\n");
	for(int i = 0; i < 4; i++)
	{
		printf("\t[");
		for(int j = 0; j < 4; j++)
		{
			printf("%f,", C[i*4+j]);
		}
		printf("],\n");
	}
	printf("]\n\n");

	hipFree(C_d);
	hipFree(B_d);
	hipFree(A_d);

	free(C);
}*/
