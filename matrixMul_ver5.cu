#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "genmatrix.h"

#include <hip/hip_runtime.h>

template <int BLOCK_SIZE, int threadElemsPerDim> __global__ void matrixMulSharedMemPrefetchMultipleElements(float *C, float *A, float *B, int width) // sprawdzi� czemu to nie dzia�a
{
	int a_start = width * BLOCK_SIZE * threadElemsPerDim * blockIdx.y, a_offset, // pocz�tek wiersza z A przez kt�ry b�dziemy si� przeiterowywa�
		b_start = BLOCK_SIZE * threadElemsPerDim * blockIdx.x, b_offset;		 // pocz�tek kolumny z B przez kt�r� b�dziemy si� przeiterowywa�

	__shared__ float A_shared[BLOCK_SIZE*threadElemsPerDim*BLOCK_SIZE*threadElemsPerDim];
	__shared__ float B_shared[BLOCK_SIZE*threadElemsPerDim*BLOCK_SIZE*threadElemsPerDim];

	//float C_local = 0.0f;
	float C_local[threadElemsPerDim*threadElemsPerDim];

	float a_prefetched[threadElemsPerDim*threadElemsPerDim],
			b_prefetched[threadElemsPerDim*threadElemsPerDim];

	int row, col;

	for(row = 0; row < threadElemsPerDim; row++)
	{
		for(col = 0; col < threadElemsPerDim; col++)
		{
			a_prefetched[row*threadElemsPerDim+col] = A[a_start + (threadIdx.y + row) * width + threadIdx.x + col];
			b_prefetched[row*threadElemsPerDim+col] = B[b_start + (threadIdx.y + row) * width + threadIdx.x + col];
			C_local[row*threadElemsPerDim+col] = 0.0f;
		}
	}
	// up: domniemanie poprawno�ci

	for(int index = 0; index < gridDim.x;) // r�wnie dobrze mog�oby by� gridDim.y bo s� r�wne
	{
		++index;

		a_offset = index * BLOCK_SIZE * threadElemsPerDim;
		b_offset = index * BLOCK_SIZE * threadElemsPerDim * width;

		// <ok>
		for(row = 0; row < threadElemsPerDim; row++)
		{
			for(col = 0; col < threadElemsPerDim; col++)
			{
				A_shared[(threadIdx.y + row) * blockDim.x * threadElemsPerDim + threadIdx.x + col] = a_prefetched[row*threadElemsPerDim+col];
				B_shared[(threadIdx.y + row) * blockDim.x * threadElemsPerDim + threadIdx.x + col] = b_prefetched[row*threadElemsPerDim+col];
			}
		}
		// </ok>

		__syncthreads(); // bariera synchronizacyjna, czekamy a� wszystkie w�tki w bloku wype�ni� pami�� wsp�dzielon�

		for(row = 0; row < threadElemsPerDim; row++)
		{
			for(col = 0; col < threadElemsPerDim; col++)
			{
				if(index < gridDim.x)
				{
					a_prefetched[row*threadElemsPerDim+col] = A[a_start + a_offset + (threadIdx.y + row) * width + threadIdx.x + col];
					b_prefetched[row*threadElemsPerDim+col] = B[b_start + b_offset + (threadIdx.y + row) * width + threadIdx.x + col];
				}
			
				for(int k = 0; k < BLOCK_SIZE*threadElemsPerDim; k++)
				{
					C_local[row*threadElemsPerDim+col] += A_shared[(threadIdx.y + row) * BLOCK_SIZE * threadElemsPerDim + k] * B_shared[k * BLOCK_SIZE * threadElemsPerDim + threadIdx.x + col];
				}
			}
		}

		__syncthreads(); // bariera synchronizacyjna, czekamy a� wszystkie w�tki w bloku oblicz� wynik cz�stkowy

		if(index * BLOCK_SIZE * threadElemsPerDim >= width)
			break;
	}

	int c_start = blockIdx.y * width * BLOCK_SIZE * threadElemsPerDim,
		c_offset = blockIdx.x * BLOCK_SIZE * threadElemsPerDim;
	for(row = 0; row < threadElemsPerDim; row++)
	{
		for(col = 0; col < threadElemsPerDim; col++)
		{
			C[c_start + c_offset + width * (threadIdx.y + row) + threadIdx.x + col] = C_local[row*threadElemsPerDim+col];
		}
	}
}

static float totalTime = 0.0f;
#define THREAD_ELEMENTS_PER_DIM 2

int performImprovedSharedMemMultipleElemsTest(dim3 block_size, int width)
{
	hipError_t error;

	float *A = (float*)malloc(width*width*sizeof(float));
	float *B = (float*)malloc(width*width*sizeof(float));

	generateTestMatrix(A, width);
	generateTestMatrix(B, width);
	
	float *C = (float*)malloc(width*width*sizeof(float));
	memset(C, 0, width*width*sizeof(float));

	float *A_d, *B_d, *C_d;

	error = hipMalloc((void**)&A_d, width*width*sizeof(float));

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not allocate memory on the device for matrix A: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMalloc((void**)&B_d, width*width*sizeof(float));

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not allocate memory on the device for matrix B: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMalloc((void**)&C_d, width*width*sizeof(float));

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not allocate memory on the device for matrix C: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMemcpy(A_d, A, width*width*sizeof(float), hipMemcpyHostToDevice);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not copy data from host to device: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipMemcpy(B_d, B, width*width*sizeof(float), hipMemcpyHostToDevice);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not copy data from host to device: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
    }

	error = hipEventRecord(start, NULL);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not record start event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	int grid_side = (int)ceil((float)width/(float)block_size.x/(float)THREAD_ELEMENTS_PER_DIM);

	for(int current_test = 0; current_test < TEST_COUNT; current_test++)
	{
		switch(block_size.x)
		{
			case 8:
				matrixMulSharedMemPrefetchMultipleElements<8, THREAD_ELEMENTS_PER_DIM><<<dim3(grid_side, grid_side), block_size>>>(C_d, A_d, B_d, width);
			break;
			case 16:
				matrixMulSharedMemPrefetchMultipleElements<16, THREAD_ELEMENTS_PER_DIM><<<dim3(grid_side, grid_side), block_size>>>(C_d, A_d, B_d, width);
			break;
			case 22:
				matrixMulSharedMemPrefetchMultipleElements<22, THREAD_ELEMENTS_PER_DIM><<<dim3(grid_side, grid_side), block_size>>>(C_d, A_d, B_d, width);
			break;
			case 32:
				matrixMulSharedMemPrefetchMultipleElements<32, THREAD_ELEMENTS_PER_DIM><<<dim3(grid_side, grid_side), block_size>>>(C_d, A_d, B_d, width);
			break;
		}

		hipDeviceSynchronize();
	}

	error = hipEventRecord(stop, NULL);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not record stop event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	error = hipEventSynchronize(stop);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not synchronize with stop event: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
	}
	
	totalTime = 0.0f;
	error = hipEventElapsedTime(&totalTime, start, stop);

	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not calculate elapsed time: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
        return -1;
	}

	float msecPerMatrixMul = totalTime / (float)TEST_COUNT;
    double flopsPerMatrixMul = 2.0 * (double)width * (double)width * (double)width;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);

	printf("%dx%d\t%dx%d\t%dx%d\t%dx%d\t%.3f\t%.2f\n", width, width, block_size.x * THREAD_ELEMENTS_PER_DIM, block_size.y * THREAD_ELEMENTS_PER_DIM, block_size.x, block_size.y, grid_side, grid_side, msecPerMatrixMul, gigaFlops);

	error = hipMemcpy(C, C_d, width*width*sizeof(float), hipMemcpyDeviceToHost);
	
	if(error != hipSuccess)
	{
		fprintf(stderr, "Could not copy data from device to host: %s (line: %d)\n", hipGetErrorString(error), __LINE__);
		return -1;
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(C_d);
	hipFree(B_d);
	hipFree(A_d);

	free(C);
	free(B);
	free(A);

	hipDeviceReset();

	return 0;
}

void performImprovedSharedMemMultipleElemsTests(void)
{
	srand((unsigned int)time(NULL));

	dim3 blockSizes[] = { dim3(8/THREAD_ELEMENTS_PER_DIM,8/THREAD_ELEMENTS_PER_DIM), dim3(16/THREAD_ELEMENTS_PER_DIM,16/THREAD_ELEMENTS_PER_DIM), dim3(22/THREAD_ELEMENTS_PER_DIM,22/THREAD_ELEMENTS_PER_DIM), dim3(32/THREAD_ELEMENTS_PER_DIM,32/THREAD_ELEMENTS_PER_DIM) };
	int matrixSizes[] = { 32, 64, 128 };

	for(int i = 0; i < sizeof(matrixSizes)/sizeof(int); i++)
	{
		for(int j = 0; j < sizeof(blockSizes)/sizeof(dim3); j++)
		{
			performImprovedSharedMemMultipleElemsTest(blockSizes[j], matrixSizes[i]);
		}
	}

	hipDeviceReset();
}
